#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Mårten Björkman aka Celebrandil //
//********************************************************//

#include <cstdio>
#include <cstring>
#include <cmath>
#include <iostream>
#include <cudautils.h>

#include "cudaImage.h"
#include "cudaSift.h"
#include "cudaSiftD.h"
#include "cudaSiftH.h"

#include "cudaSiftD.cu"
/**
 * cuda初始化
 * @param devNum 调用的设备号
 */
void InitCuda(int devNum)
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (!nDevices) {
    std::cerr << "No CUDA devices available" << std::endl;
    return;
  }
  devNum = std::min(nDevices-1, devNum);
  deviceInit(devNum); // 初始化设备，见cudaimage.cu

  // 输出调用的设备信息
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devNum);
  printf("Device Number: %d\n", devNum);
  printf("  Device name: %s\n", prop.name);
  printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1000);
  printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  printf("  Peak Memory Bandwidth (GB/s): %.1f\n\n",
	 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
}

/**
 * 提取Sift特征
 * @param siftData    sift特征点数据
 * @param img         待检测图像
 * @param numOctaves  金字塔层数
 * @param initBlur
 * @param thresh
 * @param lowestScale
 * @param subsampling
 */
void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling)
{
  TimerGPU timer(0);

  int totPts = 0;
  // 将SiftData中的可用特征点数（numPts）全局内存，分配的特征点数（maxPts）拷贝到常量内存
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

  // 金字塔参数初始化，层数，数据大小等
  const int nd = NUM_SCALES + 3; // 高斯金字塔每组需要的图像层数，见cduaSiftD.h
  int w = img.width;
  int h = img.height;
  int p = iAlignUp(w, 128); // 向上取整，用于内存对齐，加速访问
  int size = 0;         // image sizes
  int sizeTmp = nd*h*p; // 金字塔数据大小
  for (int i=0;i<numOctaves;i++) {
    w /= 2;
    h /= 2;
    int p = iAlignUp(w, 128);
    size += h*p;
    sizeTmp += nd*h*p;
  }

  float *memoryTmp = NULL;
  size_t pitch;
  size += sizeTmp;
  safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));
  float *memorySub = memoryTmp + sizeTmp;

  // 循环处理每一层，完成降采样
  ExtractSiftLoop(siftData, img, numOctaves, initBlur, thresh, lowestScale, subsampling, memoryTmp, memorySub);

  // 从全局内存中获取提前到的可用sift特征点数
  safeCall(hipMemcpyFromSymbol(&siftData.numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
  safeCall(hipFree(memoryTmp));

  // 将数据从设备端拷贝到主机端
#ifdef MANAGEDMEM
  safeCall(hipDeviceSynchronize());
#else
  if (siftData.h_data)
    safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint)*siftData.numPts, hipMemcpyDeviceToHost));
#endif
  double totTime = timer.read();
#ifndef VERBOSE
  printf("Total time incl memory =      %.2f ms\n", totTime);
#endif
}

extern double DynamicMain(CudaImage &img, SiftData &siftData, int numOctaves, double initBlur, float thresh, float lowestScale, float edgeLimit, float *memoryTmp);

/**
 * 循环处理每一层图像
 * @param siftData    siftData特征点数据
 * @param img         给定的原始图像
 * @param numOctaves  高斯金字塔层数
 * @param initBlur
 * @param thresh
 * @param lowestScale
 * @param subsampling
 * @param memoryTmp
 * @param memorySub
 */
void ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp, float *memorySub)
{
  TimerGPU timer(0);
#if 1
  int w = img.width;
  int h = img.height;
  if (numOctaves>1) {
    CudaImage subImg;
    int p = iAlignUp(w/2, 128);
    subImg.Allocate(w/2, h/2, p, false, memorySub);
    ScaleDown(subImg, img, 0.5f);
    float totInitBlur = (float)sqrt(initBlur*initBlur + 0.5f*0.5f) / 2.0f;
    ExtractSiftLoop(siftData, subImg, numOctaves-1, totInitBlur, thresh, lowestScale, subsampling*2.0f, memoryTmp, memorySub + (h/2)*p);
  }
  if (lowestScale<subsampling*2.0f)
    ExtractSiftOctave(siftData, img, initBlur, thresh, lowestScale, subsampling, memoryTmp);
#else
  DynamicMain(img, siftData, numOctaves, initBlur, thresh, lowestScale, 10.0f, memoryTmp);
#endif
  double totTime = timer.read();
#ifdef VERBOSE
  printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, double initBlur, float thresh, float lowestScale, float subsampling, float *memoryTmp)
{
  const int nd = NUM_SCALES + 3;
  TimerGPU timer0;
  CudaImage diffImg[nd];
  int w = img.width;
  int h = img.height;
  int p = iAlignUp(w, 128);
  for (int i=0;i<nd-1;i++)
    diffImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h);

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = img.d_data;
  resDesc.res.pitch2D.width = img.width;
  resDesc.res.pitch2D.height = img.height;
  resDesc.res.pitch2D.pitchInBytes = img.pitch*sizeof(float);
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0]   = hipAddressModeClamp;
  texDesc.addressMode[1]   = hipAddressModeClamp;
  texDesc.filterMode       = hipFilterModeLinear;
  texDesc.readMode         = hipReadModeElementType;
  texDesc.normalizedCoords = 0;
  // Create texture object
  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  TimerGPU timer1;
  float baseBlur = pow(2.0f, -1.0f/NUM_SCALES);
  float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
  LaplaceMulti(texObj, diffImg, baseBlur, diffScale, initBlur);
  int fstPts = 0;
  safeCall(hipMemcpyFromSymbol(&fstPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  double sigma = baseBlur*diffScale;
  FindPointsMulti(diffImg, siftData, thresh, 10.0f, sigma, 1.0f/NUM_SCALES, lowestScale/subsampling, subsampling);
  double gpuTimeDoG = timer1.read();
  TimerGPU timer4;
  int totPts = 0;
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>fstPts) {
    ComputeOrientations(texObj, siftData, fstPts, totPts);
    safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
    ExtractSiftDescriptors(texObj, siftData, fstPts, totPts, subsampling);
  }
  safeCall(hipDestroyTextureObject(texObj));
  double gpuTimeSift = timer4.read();

  double totTime = timer0.read();
#ifdef VERBOSE
  printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
  if (totPts>0)
    printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts);
#endif
}

/**
 * SiftData数据初始化
 * @param data Sift 特征点数据
 * @param num  分配的特征点个数
 * @param host 数据是否存储在主机端
 * @param dev  数据是否存储在设备端
 */
void InitSiftData(SiftData &data, int num, bool host, bool dev)
{
  data.numPts = 0;
  data.maxPts = num;
  int sz = sizeof(SiftPoint)*num;
#ifdef MANAGEDMEM
  safeCall(hipMallocManaged((void **)&data.m_data, sz)); // 分配统一内存地址
#else
  data.h_data = NULL;
  if (host)
    data.h_data = (SiftPoint *)malloc(sz);
  data.d_data = NULL;
  if (dev)
    safeCall(hipMalloc((void **)&data.d_data, sz));
#endif
}

/**
 * SiftData数据释放
 * @param data Sift 特征点数据
 */
void FreeSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  safeCall(hipFree(data.m_data));
#else
  if (data.d_data!=NULL)
    safeCall(hipFree(data.d_data));
  data.d_data = NULL;
  if (data.h_data!=NULL)
    free(data.h_data);
#endif
  data.numPts = 0;
  data.maxPts = 0;
}

/**
 * 打印Sift 特征点
 * @param data Sift 特征点数据
 */
void PrintSiftData(SiftData &data)
{
#ifdef MANAGEDMEM
  SiftPoint *h_data = data.m_data;
#else
  SiftPoint *h_data = data.h_data;
  if (data.h_data==NULL) {
    h_data = (SiftPoint *)malloc(sizeof(SiftPoint)*data.maxPts);
    safeCall(hipHostMalloc((void **)&h_data, sizeof(SiftPoint)*data.maxPts));
    safeCall(hipMemcpy(h_data, data.d_data, sizeof(SiftPoint)*data.numPts, hipMemcpyDeviceToHost));
    data.h_data = h_data;
  }
#endif
  for (int i=0;i<data.numPts;i++) {
    printf("xpos         = %.2f\n", h_data[i].xpos);
    printf("ypos         = %.2f\n", h_data[i].ypos);
    printf("scale        = %.2f\n", h_data[i].scale);
    printf("sharpness    = %.2f\n", h_data[i].sharpness);
    printf("edgeness     = %.2f\n", h_data[i].edgeness);
    printf("orientation  = %.2f\n", h_data[i].orientation);
    printf("score        = %.2f\n", h_data[i].score);
    float *siftData = (float*)&h_data[i].data;
    for (int j=0;j<8;j++) {
      if (j==0)
	printf("data = ");
      else
	printf("       ");
      for (int k=0;k<16;k++)
	if (siftData[j+8*k]<0.05)
	  printf(" .   ");
	else
	  printf("%.2f ", siftData[j+8*k]);
      printf("\n");
    }
  }
  printf("Number of available points: %d\n", data.numPts);
  printf("Number of allocated points: %d\n", data.maxPts);
}

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

/**
 * 图像降采样，并同时进行高斯滤波
 * @param  res      降采样后的图像
 * @param  src      原始图像
 * @param  variance 高斯滤波核方差
 * @return
 */
double ScaleDown(CudaImage &res, CudaImage &src, float variance)
{
  if (res.d_data==NULL || src.d_data==NULL) {
    printf("ScaleDown: missing data\n");
    return 0.0;
  }

  // 计算高斯核，1维
  float h_Kernel[5];
  float kernelSum = 0.0f;
  for (int j=0;j<5;j++) {
    h_Kernel[j] = (float)expf(-(double)(j-2)*(j-2)/2.0/variance);
    kernelSum += h_Kernel[j];
  }

  // 高斯核归一化
  for (int j=0;j<5;j++)
    h_Kernel[j] /= kernelSum;

  // 将高斯核从主机端拷贝到设备端的常量内存
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel1), h_Kernel, 5*sizeof(float)));

  // 计算线程块和线程格，注意：每一个线程块分配的线程数远小于对应的图像区域大小
  // (SCALEDOWN_W, SCALEDOWN_H)的图像块对应的线程数为(SCALEDOWN_W + 4，1)
  dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
  dim3 threads(SCALEDOWN_W + 4); // 1维高斯滤波核的大小是5，两侧各多2，所以+4

  ScaleDown<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);

  checkMsg("ScaleDown() execution failed\n");
  return 0.0;
}

double ComputeOrientations(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts)
{
  dim3 blocks(totPts - fstPts);
  dim3 threads(128);
#ifdef MANAGEDMEM
  ComputeOrientations<<<blocks, threads>>>(texObj, siftData.m_data, fstPts);
#else
  ComputeOrientations<<<blocks, threads>>>(texObj, siftData.d_data, fstPts);
#endif
  checkMsg("ComputeOrientations() execution failed\n");
  return 0.0;
}

double ExtractSiftDescriptors(hipTextureObject_t texObj, SiftData &siftData, int fstPts, int totPts, float subsampling)
{
  dim3 blocks(totPts - fstPts);
  dim3 threads(16, 8);
#ifdef MANAGEDMEM
  ExtractSiftDescriptors<<<blocks, threads>>>(texObj, siftData.m_data, fstPts, subsampling);
#else
  ExtractSiftDescriptors<<<blocks, threads>>>(texObj, siftData.d_data, fstPts, subsampling);
#endif
  checkMsg("ExtractSiftDescriptors() execution failed\n");
  return 0.0;
}

//==================== Multi-scale functions ===================//

double LaplaceMulti(hipTextureObject_t texObj, CudaImage *results, float baseBlur, float diffScale, float initBlur)
{
  float kernel[12*16];
  float scale = baseBlur;
  for (int i=0;i<NUM_SCALES+3;i++) {
    float kernelSum = 0.0f;
    float var = scale*scale - initBlur*initBlur;
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) {
      kernel[16*i+j+LAPLACE_R] = (float)expf(-(double)j*j/2.0/var);
      kernelSum += kernel[16*i+j+LAPLACE_R];
    }
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++)
      kernel[16*i+j+LAPLACE_R] /= kernelSum;
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel2), kernel, 12*16*sizeof(float)));
  int width = results[0].width;
  int pitch = results[0].pitch;
  int height = results[0].height;
  dim3 blocks(iDivUp(width+2*LAPLACE_R, LAPLACE_W), height);
  dim3 threads(LAPLACE_W+2*LAPLACE_R, LAPLACE_S);
  LaplaceMulti<<<blocks, threads>>>(texObj, results[0].d_data, width, pitch, height);
  checkMsg("LaplaceMulti() execution failed\n");
  return 0.0;
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit, float scale, float factor, float lowestScale, float subsampling)
{
  if (sources->d_data==NULL) {
    printf("FindPointsMulti: missing data\n");
    return 0.0;
  }
  int w = sources->width;
  int p = sources->pitch;
  int h = sources->height;
  float threshs[2] = { thresh, -thresh };
  float scales[NUM_SCALES];
  float diffScale = pow(2.0f, factor);
  for (int i=0;i<NUM_SCALES;i++) {
    scales[i] = scale;
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &threshs, 2*sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edgeLimit, sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, sizeof(float)*NUM_SCALES));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float)));

  dim3 blocks(iDivUp(w, MINMAX_W)*NUM_SCALES, iDivUp(h, MINMAX_H));
  dim3 threads(MINMAX_W + 2);
#ifdef MANAGEDMEM
  FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.m_data, w, p, h, NUM_SCALES, subsampling);
#else
  FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.d_data, w, p, h, NUM_SCALES, subsampling);
#endif
  checkMsg("FindPointsMulti() execution failed\n");
  return 0.0;
}

