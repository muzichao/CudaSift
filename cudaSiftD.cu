#include "hip/hip_runtime.h"
//********************************************************//
// CUDA SIFT extractor by Marten Bjorkman aka Celebrandil //
//********************************************************//

#include <cudautils.h>
#include "cudaSiftD.h"
#include "cudaSift.h"

///////////////////////////////////////////////////////////////////////////////
// Kernel configuration
///////////////////////////////////////////////////////////////////////////////

__constant__ float d_Threshold[2];
__constant__ float d_Scales[8], d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;//SiftData分配的特征点数（maxPts）- 常量内存

__device__ unsigned int d_PointCounter[1];//SiftData中的可用特征点数（numPts）- 全局内存
__constant__ float d_Kernel1[5]; // 降采样时使用的1维高斯核 - 常量内存
__constant__ float d_Kernel2[12*16]; // 设备端的多尺度1维高斯核集合 - 常量内存

///////////////////////////////////////////////////////////////////////////////
// Lowpass filter an subsample image
///////////////////////////////////////////////////////////////////////////////
/**
 * 设备端的降采样
 * @param d_Result 降采样后的图像数据
 * @param d_Data   原始图像
 * @param width    原始图像的宽度
 * @param pitch    原始图像内存对齐宽度
 * @param height   原始图像的高度
 * @param newpitch 降采样后图像内存对齐宽度
 */
__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
  __shared__ float inrow[SCALEDOWN_W + 4]; // 保存原始图像中的一行
  __shared__ float brow[5 * (SCALEDOWN_W / 2)]; // 保存按行高斯滤波后的结果，共5行
  __shared__ int yRead[SCALEDOWN_H + 4]; // 每个线程块对应原始图像的行索引的位置
  __shared__ int yWrite[SCALEDOWN_H + 4]; // 每个线程块对应降采样后的图像的行索引的位置
  #define dx2 (SCALEDOWN_W / 2)
  const int tx = threadIdx.x; // 当前线程在线程块中的列索引
  const int tx0 = tx + 0 * dx2; // 进行列滤波时的第一行索引
  const int tx1 = tx + 1 * dx2; // 进行列滤波时的第二行索引
  const int tx2 = tx + 2 * dx2; // 进行列滤波时的第三行索引
  const int tx3 = tx + 3 * dx2; // 进行列滤波时的第四行索引
  const int tx4 = tx + 4 * dx2; // 进行列滤波时的第五行索引
  const int xStart = blockIdx.x * SCALEDOWN_W; // 每个线程块对应原始图像的列索引的起始位置
  const int yStart = blockIdx.y * SCALEDOWN_H; // 每个线程块对应原始图像的行索引的起始位置
  const int xWrite = xStart / 2 + tx; // 保存数据的全局列索引
  const float *k = d_Kernel1; // 高斯核


  // 计算每个线程块对应的原始图像和降采样后的图像的行索引，并进行边界处理
  if (tx < SCALEDOWN_H + 4) {
    ///****** 是否有错误           ***********///
    ///****** y = yStart + tx - 2 **********///
    int y = yStart + tx - 1;
    y = (y < 0 ? 0 : y);
    y = (y >= height ? height - 1 : y);
    yRead[tx] = y * pitch; // 原始图像行索引
    yWrite[tx] = (yStart + tx - 4) / 2 * newpitch; // 降采样后图像行索引
  }
  __syncthreads();

  // 计算当前线程对应的列索引
  int xRead = xStart + tx - 2;
  xRead = (xRead < 0 ? 0 : xRead);
  xRead = (xRead >= width ? width - 1 : xRead);

  // 每5行（高斯核的大小）一起处理，直到处理完所有行
  for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {
    // 读原始图像的一行（5行中的第1行）
    inrow[tx] = d_Data[yRead[dy+0] + xRead];
    __syncthreads();

    // 进行行降采样及行高斯滤波（5行中的第1行）
    if (tx < dx2)
      brow[tx0] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
    __syncthreads();

    // 行滤波5行之后再进行列降采样和列高斯滤波（dy+0 = 10）
    // dy = 10, dy + 0 = 10，tx* (9, 5, 6, 7, 8)
    if (tx < dx2 && dy >= 4 && !(dy&1))
      d_Result[yWrite[dy+0] + xWrite] = k[2]*brow[tx2] + k[0]*(brow[tx0]+brow[tx4]) + k[1]*(brow[tx1]+brow[tx3]);

    if (dy < (SCALEDOWN_H + 3)) {
      // 读原始图像的一行（5行中的第2行）
      inrow[tx] = d_Data[yRead[dy+1] + xRead];
      __syncthreads();

      // 进行行降采样及行高斯滤波（5行中的第2行）
      if (tx < dx2)
      	brow[tx1] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();

      // 行滤波5行之后再进行列降采样和列高斯滤波
      // dy = 5, dy + 1 = 6，tx* (4, 5, 1, 2, 3)
      // dy = 15, dy + 1 = 16, tx* (14, 15, 11, 12, 13)
      if (tx < dx2 && dy >= 3 && (dy&1))
        d_Result[yWrite[dy+1] + xWrite] = k[2]*brow[tx3] + k[0]*(brow[tx1]+brow[tx0]) + k[1]*(brow[tx2]+brow[tx4]);
    }

    if (dy < (SCALEDOWN_H + 2)) {
      // 读原始图像的一行（5行中的第3行）
      inrow[tx] = d_Data[yRead[dy+2] + xRead];
      __syncthreads();

      // 进行行降采样及行高斯滤波（5行中的第3行）
      if (tx < dx2)
        brow[tx2] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();

      // 行滤波5行之后再进行列降采样和列高斯滤波
      // dy = 10, dy + 2 = 12，tx* (9, 10, 11, 7, 8)
      if (tx < dx2 && dy >= 2 && !(dy&1))
        d_Result[yWrite[dy+2] + xWrite] = k[2]*brow[tx4] + k[0]*(brow[tx2]+brow[tx1]) + k[1]*(brow[tx3]+brow[tx0]);
    }

    if (dy < (SCALEDOWN_H + 1)) {
      // 读原始图像的一行（5行中的第4行）
      inrow[tx] = d_Data[yRead[dy+3] + xRead];
      __syncthreads();

      // 进行行降采样及行高斯滤波（5行中的第4行）
      if (tx < dx2)
        brow[tx3] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();

      // 行滤波5行之后再进行列降采样和列高斯滤波
      // dy = 5, dy + 3 = 8，tx* (4, 5, 6, 7, 3)
      // dy = 15, dy + 3 = 18, tx* (14, 15, 16, 17, 13)
      if (tx < dx2 && dy >= 1 && (dy&1))
        d_Result[yWrite[dy+3] + xWrite] = k[2]*brow[tx0] + k[0]*(brow[tx3]+brow[tx2]) + k[1]*(brow[tx4]+brow[tx1]);
    }

    if (dy < SCALEDOWN_H) {
      // 读原始图像的一行（5行中的第5行）
      inrow[tx] = d_Data[yRead[dy+4] + xRead];
      __syncthreads();

      // 进行行降采样及行高斯滤波（5行中的第5行）
      if (tx<dx2)
        brow[tx4] = k[0]*(inrow[2*tx]+inrow[2*tx+4]) + k[1]*(inrow[2*tx+1]+inrow[2*tx+3]) + k[2]*inrow[2*tx+2];
      __syncthreads();

      // 行滤波5行之后再进行列降采样和列高斯滤波
      // dy = 0, dy + 4 = 4，tx* (0, 0, 1, 2, 3)
      // dy = 10, dy + 4 = 14, tx* (9, 10, 11, 12, 13)
      if (tx<dx2 && !(dy&1))
        d_Result[yWrite[dy+4] + xWrite] = k[2]*brow[tx1] + k[0]*(brow[tx4]+brow[tx3]) + k[1]*(brow[tx0]+brow[tx2]);
    }

    __syncthreads();
  }
}


__global__ void ExtractSiftDescriptors(hipTextureObject_t texObj, SiftPoint *d_sift, int fstPts, float subsampling)
{
  __shared__ float gauss[16];
  __shared__ float buffer[128];
  __shared__ float sums[128];

  const int tx = threadIdx.x; // 0 -> 16
  const int ty = threadIdx.y; // 0 -> 16
  const int idx = ty*16 + tx;
  const int bx = blockIdx.x + fstPts;  // 0 -> numPts
  if (ty==0)
    gauss[tx] = exp(-(tx-7.5f)*(tx-7.5f)/128.0f);
  buffer[idx] = 0.0f;
  __syncthreads();

  // Compute angles and gradients
  float theta = 2.0f*3.1415f/360.0f*d_sift[bx].orientation;
  float sina = sinf(theta);           // cosa -sina
  float cosa = cosf(theta);           // sina  cosa
  float scale = 12.0f/16.0f*d_sift[bx].scale;
  float ssina = scale*sina;
  float scosa = scale*cosa;

  for (int y=ty;y<16;y+=8) {
    float xpos = d_sift[bx].xpos + (tx-7.5f)*scosa - (y-7.5f)*ssina;
    float ypos = d_sift[bx].ypos + (tx-7.5f)*ssina + (y-7.5f)*scosa;
    float dx = tex2D<float>(texObj, xpos+cosa, ypos+sina) -
      tex2D<float>(texObj, xpos-cosa, ypos-sina);
    float dy = tex2D<float>(texObj, xpos-sina, ypos+cosa) -
      tex2D<float>(texObj, xpos+sina, ypos-cosa);
    float grad = gauss[y]*gauss[tx] * sqrtf(dx*dx + dy*dy);
    float angf = 4.0f/3.1415f*atan2f(dy, dx) + 4.0f;

    int hori = (tx + 2)/4 - 1;      // Convert from (tx,y,angle) to bins
    float horf = (tx - 1.5f)/4.0f - hori;
    float ihorf = 1.0f - horf;
    int veri = (y + 2)/4 - 1;
    float verf = (y - 1.5f)/4.0f - veri;
    float iverf = 1.0f - verf;
    int angi = angf;
    int angp = (angi<7 ? angi+1 : 0);
    angf -= angi;
    float iangf = 1.0f - angf;

    int hist = 8*(4*veri + hori);   // Each gradient measure is interpolated
    int p1 = angi + hist;           // in angles, xpos and ypos -> 8 stores
    int p2 = angp + hist;
    if (tx>=2) {
      float grad1 = ihorf*grad;
      if (y>=2) {   // Upper left
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1, iangf*grad2);
	atomicAdd(buffer + p2,  angf*grad2);
      }
      if (y<=13) {  // Lower left
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+32, iangf*grad2);
	atomicAdd(buffer + p2+32,  angf*grad2);
      }
    }
    if (tx<=14) {
      float grad1 = horf*grad;
      if (y>=2) {    // Upper right
        float grad2 = iverf*grad1;
	atomicAdd(buffer + p1+8, iangf*grad2);
	atomicAdd(buffer + p2+8,  angf*grad2);
      }
      if (y<=13) {   // Lower right
        float grad2 = verf*grad1;
	atomicAdd(buffer + p1+40, iangf*grad2);
	atomicAdd(buffer + p2+40,  angf*grad2);
      }
    }
  }
  __syncthreads();

  // Normalize twice and suppress peaks first time
  if (idx<64)
    sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
  __syncthreads();
  if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
  __syncthreads();
  if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
  __syncthreads();
  if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
  __syncthreads();
  if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
  __syncthreads();
  float tsum1 = sums[0] + sums[1] + sums[2] + sums[3];
  buffer[idx] = buffer[idx] * rsqrtf(tsum1);

  if (buffer[idx]>0.2f)
    buffer[idx] = 0.2f;
  __syncthreads();
  if (idx<64)
    sums[idx] = buffer[idx]*buffer[idx] + buffer[idx+64]*buffer[idx+64];
  __syncthreads();
  if (idx<32) sums[idx] = sums[idx] + sums[idx+32];
  __syncthreads();
  if (idx<16) sums[idx] = sums[idx] + sums[idx+16];
  __syncthreads();
  if (idx<8)  sums[idx] = sums[idx] + sums[idx+8];
  __syncthreads();
  if (idx<4)  sums[idx] = sums[idx] + sums[idx+4];
  __syncthreads();
  float tsum2 = sums[0] + sums[1] + sums[2] + sums[3];

  float *desc = d_sift[bx].data;
  desc[idx] = buffer[idx] * rsqrtf(tsum2);
  if (idx==0) {
    d_sift[bx].xpos *= subsampling;
    d_sift[bx].ypos *= subsampling;
    d_sift[bx].scale *= subsampling;
  }
}


__global__ void ComputeOrientations(hipTextureObject_t texObj, SiftPoint *d_Sift, int fstPts)
{
  __shared__ float hist[64];
  __shared__ float gauss[11];
  const int tx = threadIdx.x;
  const int bx = blockIdx.x + fstPts;
  float i2sigma2 = -1.0f/(4.5f*d_Sift[bx].scale*d_Sift[bx].scale);
  if (tx<11)
    gauss[tx] = exp(i2sigma2*(tx-5)*(tx-5));
  if (tx<64)
    hist[tx] = 0.0f;
  __syncthreads();
  float xp = d_Sift[bx].xpos - 5.0f;
  float yp = d_Sift[bx].ypos - 5.0f;
  int yd = tx/11;
  int xd = tx - yd*11;
  float xf = xp + xd;
  float yf = yp + yd;
  if (yd<11) {
    float dx = tex2D<float>(texObj, xf+1.0, yf) - tex2D<float>(texObj, xf-1.0, yf);
    float dy = tex2D<float>(texObj, xf, yf+1.0) - tex2D<float>(texObj, xf, yf-1.0);
    int bin = 16.0f*atan2f(dy, dx)/3.1416f + 16.5f;
    if (bin>31)
      bin = 0;
    float grad = sqrtf(dx*dx + dy*dy);
    atomicAdd(&hist[bin], grad*gauss[xd]*gauss[yd]);
  }
  __syncthreads();
  int x1m = (tx>=1 ? tx-1 : tx+31);
  int x1p = (tx<=30 ? tx+1 : tx-31);
  if (tx<32) {
    int x2m = (tx>=2 ? tx-2 : tx+30);
    int x2p = (tx<=29 ? tx+2 : tx-30);
    hist[tx+32] = 6.0f*hist[tx] + 4.0f*(hist[x1m] + hist[x1p]) + (hist[x2m] + hist[x2p]);
  }
  __syncthreads();
  if (tx<32) {
    float v = hist[32+tx];
    hist[tx] = (v>hist[32+x1m] && v>=hist[32+x1p] ? v : 0.0f);
  }
  __syncthreads();
  if (tx==0) {
    float maxval1 = 0.0;
    float maxval2 = 0.0;
    int i1 = -1;
    int i2 = -1;
    for (int i=0;i<32;i++) {
      float v = hist[i];
      if (v>maxval1) {
	maxval2 = maxval1;
	maxval1 = v;
	i2 = i1;
	i1 = i;
      } else if (v>maxval2) {
	maxval2 = v;
	i2 = i;
      }
    }
    float val1 = hist[32+((i1+1)&31)];
    float val2 = hist[32+((i1+31)&31)];
    float peak = i1 + 0.5f*(val1-val2) / (2.0f*maxval1-val1-val2);
    d_Sift[bx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);
    if (maxval2>0.8f*maxval1 && false) {
      float val1 = hist[32+((i2+1)&31)];
      float val2 = hist[32+((i2+31)&31)];
      float peak = i2 + 0.5f*(val1-val2) / (2.0f*maxval2-val1-val2);
      unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
      if (idx<d_MaxNumPoints) {
	d_Sift[idx].xpos = d_Sift[bx].xpos;
	d_Sift[idx].ypos = d_Sift[bx].ypos;
	d_Sift[idx].scale = d_Sift[bx].scale;
	d_Sift[idx].sharpness = d_Sift[bx].sharpness;
	d_Sift[idx].edgeness = d_Sift[bx].edgeness;
	d_Sift[idx].orientation = 11.25f*(peak<0.0f ? peak+32.0f : peak);;
	d_Sift[idx].subsampling = d_Sift[bx].subsampling;
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////
// Subtract two images (multi-scale version)
///////////////////////////////////////////////////////////////////////////////

 __global__ void FindPointsMulti(float *d_Data0, SiftPoint *d_Sift, int width, int pitch, int height, int nScales, float subsampling)
{
  #define MEMWID (MINMAX_W + 2)  // (128)
  __shared__ float ymin1[MEMWID], ymin2[MEMWID], ymin3[MEMWID];
  __shared__ float ymax1[MEMWID], ymax2[MEMWID], ymax3[MEMWID];
  __shared__ unsigned int cnt;
  __shared__ unsigned short points[96];

  int tx = threadIdx.x; // 当前线程的列索引
  int block = blockIdx.x / nScales; // 当前线程对应的 nScales，中间组数
  int scale = blockIdx.x - nScales * block; // 当前线程对应某个中间组的列数
  int minx = block * MINMAX_W;
  int maxx = min(minx + MINMAX_W, width);
  int xpos = minx + tx;
  int size = pitch * height;
  int ptr = size * scale + max(min(xpos-1, width-1), 0);

  if (tx == 0)
    cnt = 0;
  __syncthreads();

  int yloops = min(height - MINMAX_H * blockIdx.y, MINMAX_H);
  for (int y=0; y < yloops; y++) {

    int ypos = MINMAX_H * blockIdx.y + y;
    int yptr0 = ptr + max(0, ypos-1) * pitch;
    int yptr1 = ptr + ypos * pitch;
    int yptr2 = ptr + min(height-1, ypos+1) * pitch;

    {
      float d10 = d_Data0[yptr0];
      float d11 = d_Data0[yptr1];
      float d12 = d_Data0[yptr2];
      ymin1[tx] = fminf(fminf(d10, d11), d12);
      ymax1[tx] = fmaxf(fmaxf(d10, d11), d12);
    }

    {
      float d30 = d_Data0[yptr0 + 2*size];
      float d31 = d_Data0[yptr1 + 2*size];
      float d32 = d_Data0[yptr2 + 2*size];
      ymin3[tx] = fminf(fminf(d30, d31), d32);
      ymax3[tx] = fmaxf(fmaxf(d30, d31), d32);
    }

    float d20 = d_Data0[yptr0 + 1*size];
    float d21 = d_Data0[yptr1 + 1*size];
    float d22 = d_Data0[yptr2 + 1*size];
    ymin2[tx] = fminf(fminf(ymin1[tx], fminf(fminf(d20, d21), d22)), ymin3[tx]);
    ymax2[tx] = fmaxf(fmaxf(ymax1[tx], fmaxf(fmaxf(d20, d21), d22)), ymax3[tx]);

    __syncthreads();

    if (tx > 0 && tx < MINMAX_W + 1 && xpos <= maxx) {
      if (d21 < d_Threshold[1]) {
        float minv = fminf(fminf(fminf(ymin2[tx-1], ymin2[tx+1]), ymin1[tx]), ymin3[tx]);
        minv = fminf(fminf(minv, d20), d22);
        if (d21<minv) {
          int pos = atomicInc(&cnt, 31);
          points[3*pos+0] = xpos - 1;
          points[3*pos+1] = ypos;
          points[3*pos+2] = scale;
        }
      }

      if (d21 > d_Threshold[0]) {
        float maxv = fmaxf(fmaxf(fmaxf(ymax2[tx-1], ymax2[tx+1]), ymax1[tx]), ymax3[tx]);
        maxv = fmaxf(fmaxf(maxv, d20), d22);
        if (d21>maxv) {
          int pos = atomicInc(&cnt, 31);
          points[3*pos+0] = xpos - 1;
          points[3*pos+1] = ypos;
          points[3*pos+2] = scale;
        }
      }

    }

    __syncthreads();
  }

  if (tx < cnt) {
    int xpos = points[3*tx+0];
    int ypos = points[3*tx+1];
    int scale = points[3*tx+2];
    int ptr = xpos + (ypos + (scale+1)*height)*pitch;
    float val = d_Data0[ptr];
    float *data1 = &d_Data0[ptr];
    float dxx = 2.0f*val - data1[-1] - data1[1];
    float dyy = 2.0f*val - data1[-pitch] - data1[pitch];
    float dxy = 0.25f*(data1[+pitch+1] + data1[-pitch-1] - data1[-pitch+1] - data1[+pitch-1]);
    float tra = dxx + dyy;
    float det = dxx*dyy - dxy*dxy;

    if (tra*tra<d_EdgeLimit*det) {
      float edge = __fdividef(tra*tra, det);
      float dx = 0.5f*(data1[1] - data1[-1]);
      float dy = 0.5f*(data1[pitch] - data1[-pitch]);
      float *data0 = d_Data0 + ptr - height*pitch;
      float *data2 = d_Data0 + ptr + height*pitch;
      float ds = 0.5f*(data0[0] - data2[0]);
      float dss = 2.0f*val - data2[0] - data0[0];
      float dxs = 0.25f*(data2[1] + data0[-1] - data0[1] - data2[-1]);
      float dys = 0.25f*(data2[pitch] + data0[-pitch] - data2[-pitch] - data0[pitch]);
      float idxx = dyy*dss - dys*dys;
      float idxy = dys*dxs - dxy*dss;
      float idxs = dxy*dys - dyy*dxs;
      float idet = __fdividef(1.0f, idxx*dxx + idxy*dxy + idxs*dxs);
      float idyy = dxx*dss - dxs*dxs;
      float idys = dxy*dxs - dxx*dys;
      float idss = dxx*dyy - dxy*dxy;
      float pdx = idet*(idxx*dx + idxy*dy + idxs*ds);
      float pdy = idet*(idxy*dx + idyy*dy + idys*ds);
      float pds = idet*(idxs*dx + idys*dy + idss*ds);

      if (pdx<-0.5f || pdx>0.5f || pdy<-0.5f || pdy>0.5f || pds<-0.5f || pds>0.5f) {
        pdx = __fdividef(dx, dxx);
      	pdy = __fdividef(dy, dyy);
      	pds = __fdividef(ds, dss);
      }

      float dval = 0.5f*(dx*pdx + dy*pdy + ds*pds);
      int maxPts = d_MaxNumPoints;
      unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
      idx = (idx>=maxPts ? maxPts-1 : idx);
      d_Sift[idx].xpos = xpos + pdx;
      d_Sift[idx].ypos = ypos + pdy;
      d_Sift[idx].scale = d_Scales[scale] * exp2f(pds*d_Factor);
      d_Sift[idx].sharpness = val + dval;
      d_Sift[idx].edgeness = edge;
      d_Sift[idx].subsampling = subsampling;
    }
  }
}

/**
 * 计算一层高斯金字塔和DOG金字塔
 * @param texObj   图像的纹理数据
 * @param d_Result DOG金字塔的某层
 * @param width    图像的宽度
 * @param pitch    内存对齐宽度
 * @param height   图像高度
 */
 __global__ void LaplaceMulti(hipTextureObject_t texObj, float *d_Result, int width, int pitch, int height)
{
  __shared__ float data1[(LAPLACE_W + 2 * LAPLACE_R) * LAPLACE_S];
  __shared__ float data2[LAPLACE_W * LAPLACE_S];
  const int tx = threadIdx.x; // 当前线程的列索引
  const int xp = blockIdx.x*LAPLACE_W + tx; // 当前组的列索引
  const int yp = blockIdx.y; // 每组的第几行
  const int scale = threadIdx.y; // 当前层的第几组
  float *kernel = d_Kernel2 + scale*16; // 每行线程对应组高斯核
  float *sdata1 = data1 + (LAPLACE_W + 2 * LAPLACE_R) * scale; // 每行线程对应的共享内存
  float x = xp - 3.5;
  float y = yp + 0.5;

  // 列高斯滤波
  sdata1[tx] = kernel[4] * tex2D<float>(texObj, x, y) +
    kernel[3] * (tex2D<float>(texObj, x, y-1.0) + tex2D<float>(texObj, x, y+1.0)) +
    kernel[2] * (tex2D<float>(texObj, x, y-2.0) + tex2D<float>(texObj, x, y+2.0)) +
    kernel[1] * (tex2D<float>(texObj, x, y-3.0) + tex2D<float>(texObj, x, y+3.0)) +
    kernel[0] * (tex2D<float>(texObj, x, y-4.0) + tex2D<float>(texObj, x, y+4.0));
  __syncthreads();

  float *sdata2 = data2 + LAPLACE_W * scale;

  // 行高斯滤波
  if (tx < LAPLACE_W) {
    sdata2[tx] = kernel[4]*sdata1[tx+4] +
      kernel[3] * (sdata1[tx+3] + sdata1[tx+5]) +
      kernel[2] * (sdata1[tx+2] + sdata1[tx+6]) +
      kernel[1] * (sdata1[tx+1] + sdata1[tx+7]) +
      kernel[0] * (sdata1[tx+0] + sdata1[tx+8]);
  }
  __syncthreads();

  // 差分
  if (tx < LAPLACE_W && scale < LAPLACE_S-1 && xp<width)
    d_Result[scale * height * pitch + yp * pitch + xp] = sdata2[tx] - sdata2[tx + LAPLACE_W];
}


